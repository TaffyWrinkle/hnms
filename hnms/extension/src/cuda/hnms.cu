
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA
#include "hnms.h"
#include <ATen/ATen.h>
#include <ATen/cuda/CUDAContext.h>

#include <THC/THC.h>
#include <THC/THCDeviceUtils.cuh>

#include <vector>
#include <iostream>

#include <ctime>
#include <chrono>
using namespace std::chrono;


#define CONF_TO_INT_MULT 1000000
#define CONF_TO_INT_ADD 100000
#define CONF_TO_INT(x) (long long)((x) * CONF_TO_INT_MULT) + CONF_TO_INT_ADD

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)
const int CUDA_NUM_THREADS = 512;

int const threadsPerBlock = sizeof(unsigned long long) * 8;

inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}


template <typename T>
__global__ void hnms_max_conf_kernel(long long nthreads,
        T* box_confs,
        int64_t* cell_indices,
        int64_t* cell_max_confs) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
      unsigned long long conf = CONF_TO_INT(box_confs[i]);
      unsigned long long cell = cell_indices[i];
      unsigned long long * cell_max = (unsigned long long*)(cell_max_confs + cell);
      // long long type is not supported for atomiMax
      atomicMax(cell_max, conf);
  }
}

template <typename T>
__global__ void hnms_max_idx_kernel(long long nthreads,
        T* box_confs,
        int64_t* cell_indices,
        int64_t* cell_max_confs) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {
      unsigned long long conf = CONF_TO_INT(box_confs[i]);
      auto cell = cell_indices[i];
      unsigned long long* cell_max = (unsigned long long*)(cell_max_confs + cell);
      // no implementation to take long long, but unsigned long long
      atomicCAS(cell_max, conf, (unsigned long long)i);
  }
}

template <typename T>
__global__ void hash_rects_kernel(int64_t nthreads,
        T* dets,
        T w0, T h0, T alpha,
        T bx, T by,
        T alpha_ratio,
        int64_t* out) {
    CUDA_1D_KERNEL_LOOP(idx_box, nthreads) {
        auto log_w0 = log(w0);
        auto log_h0 = log(h0);
        auto log_alpha = log(alpha);

        auto curr_det = dets + idx_box * 4;
        auto x = curr_det[0];
        auto y = curr_det[1];
        auto w = curr_det[2];
        auto h = curr_det[3];
        auto w0_alpha = w0 * alpha_ratio;
        auto h0_alpha = h0 * alpha_ratio;

        auto i = round((log_w0 - log(w)) / log_alpha);
        auto j = round((log_h0 - log(h)) / log_alpha);
        auto di = w0_alpha / pow(alpha, i);
        auto dj = h0_alpha / pow(alpha, j);

        int64_t qx, qy;
        qx = round(x / di - bx);
        qy = round(y / dj - by);
        auto curr_out  = out + 4 * idx_box;
        curr_out[0] = qx;
        curr_out[1] = qy;
        curr_out[2] = i;
        curr_out[3] = j;
    }
}

at::Tensor hash_rects_cuda(const at::Tensor& dets,
               float w0,
               float h0,
               float alpha,
               float bx,
               float by) {
    auto num_box = dets.size(0);
    auto alpha_ratio = (1. - alpha) / (1. + alpha);

    auto result = at::zeros({long(num_box), 4},
            dets.options().dtype(at::kLong));

    AT_DISPATCH_FLOATING_TYPES(dets.type(), "HASH_RECTS", [&] {
            hash_rects_kernel<scalar_t><<<GET_BLOCKS(num_box), CUDA_NUM_THREADS>>>(num_box,
                    dets.data<scalar_t>(),
                    (scalar_t)w0, (scalar_t)h0, (scalar_t)alpha,
                    (scalar_t)bx, (scalar_t)by,
                    alpha_ratio,
                    result.data<int64_t>());
            });
    return result;
}

__global__ void map_code(int num_box,
        int64_t* codes,
        int64_t* codes_as_one) {
    CUDA_1D_KERNEL_LOOP(idx_box, num_box) {
        auto curr_code = codes + 4 * idx_box;
        auto curr_mapped = codes_as_one + idx_box;
        *curr_mapped = curr_code[0] +
            curr_code[1] * 10000 +
            curr_code[2] * 100000000 +
            curr_code[3] * 1000000000000;
    }
}

at::Tensor get_best_idx_each_code(
        at::Tensor codes,
        const at::Tensor& scores) {
    auto num_box = codes.size(0);
    auto codes_as_one = at::zeros({long(num_box)},
            codes.options().dtype(at::kLong));
    map_code<<<GET_BLOCKS(num_box), CUDA_NUM_THREADS>>>(num_box,
            codes.data<int64_t>(),
            codes_as_one.data<int64_t>());
    THCudaCheck(cudaGetLastError());

    auto unique_result = at::unique_dim(codes_as_one, 0, // dim
            false, true);

    at::Tensor reverse_index = std::get<1>(unique_result);
    auto count = std::get<0>(unique_result).size(0);

    auto result = at::zeros({long(count)},
            codes.options().dtype(at::kLong));

    // get the maximum confidence score for each code with the atomic operation
    // of atomicMax.
    AT_DISPATCH_FLOATING_TYPES(scores.type(), "HNMS_MAX_IDX_KERNEL", [&] {
        hnms_max_conf_kernel<scalar_t><<<GET_BLOCKS(num_box), CUDA_NUM_THREADS>>>(
                num_box,
                scores.data<scalar_t>(),
                reverse_index.data<int64_t>(),
                result.data<int64_t>());
            });
    THCudaCheck(cudaGetLastError());

    AT_DISPATCH_FLOATING_TYPES(scores.type(), "HNMS_MAX_IDX_KERNEL", [&] {
            hnms_max_idx_kernel<scalar_t><<<GET_BLOCKS(num_box), CUDA_NUM_THREADS>>>(
                    num_box,
                    scores.data<scalar_t>(),
                    reverse_index.data_ptr<int64_t>(),
                    result.data<int64_t>());
            // NULL,
            });
    return result;
}

at::Tensor hnms_cuda(const at::Tensor& dets,
               const at::Tensor& scores,
               float w0,
               float h0,
               float alpha,
               float bx,
               float by
               ) {
    AT_ASSERTM(dets.type().is_cuda(), "dets must be a CUDA tensor");
    AT_ASSERTM(scores.type().is_cuda(), "scores must be a CUDA tensor");
    AT_ASSERTM(dets.type() == scores.type(), "dets should have the same type as scores");
    if (dets.numel() == 0) {
      return at::empty({0}, dets.options().dtype(at::kLong).device(at::kCPU));
    }

    auto codes = hash_rects_cuda(dets, w0, h0, alpha, bx, by);
    auto result = get_best_idx_each_code(codes, scores);
    return result;
}

#endif
